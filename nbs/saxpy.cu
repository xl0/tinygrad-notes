
#include <hip/hip_runtime.h>
#include <stdint.h>

// SAXPY kernel: y = alpha*x + y
__global__ void saxpy(float alpha, float *x, float *y, int32_t n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = alpha * x[i] + y[i];
    }
}
